#include "hip/hip_runtime.h"
/*
	EDGE DETECTION
	HUDSON PEDEN
	10/11/2016
*/

#include "libwb\wb.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "math.h"

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
		    }                                                                     \
      } while (0)

//@@ INSERT CODE HERE

#define TILE_WIDTH 16
#define OUTCHANNELS 3 //(Color:3 ; Grayscale: 1)

__global__ void rgb2gray(float *grayImage, float *rgbImage, int channels, int width, int height) {
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	if (x < width  && y < height){
		int offset = y * width + x;
		float greyVal = 0.0;
		
		/*R*/
		if (offset % 3 == 0) greyVal = (rgbImage[offset + 0] + rgbImage[offset + 1] + rgbImage[offset + 2]) / 3.0;

		/*G*/
		else if (offset % 3 == 1) greyVal = (rgbImage[offset - 1] + rgbImage[offset + 0] + rgbImage[offset + 1]) / 3.0;

		/*B*/
		else greyVal = (rgbImage[offset - 2] + rgbImage[offset - 1] + rgbImage[offset + 0]) / 3.0;
		
		grayImage[offset] = greyVal;
		
	}

}

int main(int argc, char *argv[]) {
	wbArg_t args;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *deviceInputImageData;
	float *deviceOutputImageData;

	args = wbArg_read(argc, argv); /* parse the input arguments */

	inputImageFile = wbArg_getInputFile(args, 0);

	inputImage = wbImport(inputImageFile); //"pict.ppm"

	imageWidth = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	// For this lab the value is always 3
	imageChannels = wbImage_getChannels(inputImage);

	// Since the image is monochromatic, it only contains one channel
	outputImage = wbImage_new(imageWidth, imageHeight, OUTCHANNELS);// 3);// 1);

	hostInputImageData = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceOutputImageData, imageWidth * imageHeight * OUTCHANNELS * sizeof(float));
	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
	wbTime_stop(Copy, "Copying data to the GPU");

	///////////////////////////////////////////////////////
	wbTime_start(Compute, "Doing the computation on the GPU");
	//@@ INSERT CODE HERE
	dim3 dimGrid(ceil((float)(imageWidth * OUTCHANNELS) / TILE_WIDTH), ceil((float)imageHeight / TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	rgb2gray <<<dimGrid, dimBlock >>>(deviceOutputImageData, deviceInputImageData, imageChannels, imageWidth * OUTCHANNELS, imageHeight);
	wbTime_stop(Compute, "Doing the computation on the GPU");

	///////////////////////////////////////////////////////
	wbTime_start(Copy, "Copying data from the GPU");
	hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * OUTCHANNELS * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying data from the GPU");
	//	for (int i = 0; i < 100; i++)fprintf(stderr, "%2.1f ", hostInputImageData[i]); fprintf(stderr, "\n");
	//	for (int i = 0; i < 100; i++)fprintf(stderr, "%2.1f ", hostOutputImageData[i]); fprintf(stderr, "\n");
	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(args, outputImage);

	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}
